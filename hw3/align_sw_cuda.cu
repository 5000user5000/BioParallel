#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <string.h>

#include "align_sw.hpp"  // AlignmentResult struct

__global__
void smith_waterman_kernel_wavefront(
    const char* seq1, const char* seq2,
    int* H,
    int m, int n,
    int match, int mismatch, int gap,
    int wave) // 現在要算第 wave 條斜線
{
    int tx = threadIdx.x;
    int ty = wave - tx;

    if (tx >= 1 && tx <= n && ty >= 1 && ty <= m) {
        int idx = ty * (n + 1) + tx;
        int idx_diag = (ty-1) * (n + 1) + (tx-1);
        int idx_up   = (ty-1) * (n + 1) + tx;
        int idx_left = ty * (n + 1) + (tx-1);

        int score_match = (seq1[ty-1] == seq2[tx-1]) ? match : mismatch;

        int score_diag = H[idx_diag] + score_match;
        int score_up = H[idx_up] + gap;
        int score_left = H[idx_left] + gap;

        int max_score = max(0, max(score_diag, max(score_up, score_left)));
        H[idx] = max_score;
    }
}


AlignmentResult smith_waterman_cuda(
    const std::string& seq1,
    const std::string& seq2,
    int match,
    int mismatch,
    int gap)
{
    size_t m = seq1.size(), n = seq2.size();
    std::vector<int> H((m+1)*(n+1), 0);

    // Allocate device memory
    char *d_seq1, *d_seq2;
    int *d_H;
    hipMalloc(&d_seq1, m * sizeof(char));
    hipMalloc(&d_seq2, n * sizeof(char));
    hipMalloc(&d_H, (m + 1) * (n + 1) * sizeof(int));

    // Copy input to device
    hipMemcpy(d_seq1, seq1.data(), m * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, seq2.data(), n * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_H, H.data(), (m + 1) * (n + 1) * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, m);

    // smith_waterman_kernel<<<gridSize, blockSize>>>(d_seq1, d_seq2, d_H, m, n, match, mismatch, gap);
    // hipDeviceSynchronize();

    // Wavefront computation
    for (int wave = 2; wave <= m + n; ++wave) { // wave從2開始，因為座標都是從1開始
        int num_threads = min(wave-1, (int)min((size_t)n, (size_t)m));
    
        if (num_threads <= 0) continue; // 防止空 kernel
    
        smith_waterman_kernel_wavefront<<<1, wave>>>(d_seq1, d_seq2, d_H, m, n, match, mismatch, gap, wave);
        hipDeviceSynchronize();
    }


    // Copy back results
    hipMemcpy(H.data(), d_H, (m + 1) * (n + 1) * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(d_H);

    // --- Find max score and position
    int max_score = 0;
    int max_i = 0, max_j = 0;
    for (size_t i = 1; i <= m; ++i) {
        for (size_t j = 1; j <= n; ++j) {
            if (H[i*(n+1) + j] > max_score) {
                max_score = H[i*(n+1) + j];
                max_i = i;
                max_j = j;
            }
        }
    }

    // --- Traceback
    std::string aligned_seq1 = "";
    std::string aligned_seq2 = "";

    int i = max_i;
    int j = max_j;

    while (i > 0 && j > 0) {
        int idx = i*(n+1) + j;
        int idx_diag = (i-1)*(n+1) + (j-1);
        int idx_up = (i-1)*(n+1) + j;
        int idx_left = i*(n+1) + (j-1);

        if (H[idx] == 0)
            break;

        if (H[idx] == H[idx_diag] + ((seq1[i-1] == seq2[j-1]) ? match : mismatch)) {
            aligned_seq1 = seq1[i-1] + aligned_seq1;
            aligned_seq2 = seq2[j-1] + aligned_seq2;
            i--;
            j--;
        } else if (H[idx] == H[idx_left] + gap) {
            aligned_seq1 = '-' + aligned_seq1;
            aligned_seq2 = seq2[j-1] + aligned_seq2;
            j--;
        } else {
            aligned_seq1 = seq1[i-1] + aligned_seq1;
            aligned_seq2 = '-' + aligned_seq2;
            i--;
        }
    }

    int start1 = i;
    int start2 = j;

    // --- Build match_line
    std::string match_line = "";
    for (size_t k = 0; k < aligned_seq1.size(); ++k) {
        if (aligned_seq1[k] == aligned_seq2[k])
            match_line += "|";
        else if (aligned_seq1[k] == '-' || aligned_seq2[k] == '-')
            match_line += " ";
        else
            match_line += "*";
    }

    // --- Return result
    AlignmentResult result;
    result.aligned_seq1 = aligned_seq1;
    result.aligned_seq2 = aligned_seq2;
    result.match_line = match_line;
    result.start1 = start1;
    result.start2 = start2;
    result.score = max_score;
    return result;
}
